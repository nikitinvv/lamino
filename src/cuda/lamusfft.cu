#include "hip/hip_runtime.h"
#include <stdio.h>

#include "lamusfft.cuh"
#include "kernels.cu"
#include "shift.cu"

lamusfft::lamusfft(size_t n0, size_t n1, size_t n2, size_t det, size_t ntheta, float phi, float gamma, float eps)
    : n0(n0), n1(n1), n2(n2), det(det), ntheta(ntheta), phi(phi), gamma(gamma) {
  mu0 = -log(eps) / (2 * n0 * n0);
  mu1 = -log(eps) / (2 * n1 * n1);
  mu2 = -log(eps) / (2 * n2 * n2);
  m0 = ceil(2 * n0 * 1 / PI * sqrt(-mu0 * log(eps) + (mu0 * n0) * (mu0 * n0) / 4));
  m1 = ceil(2 * n1 * 1 / PI * sqrt(-mu1 * log(eps) + (mu1 * n1) * (mu1 * n1) / 4));
  m2 = ceil(2 * n2 * 1 / PI * sqrt(-mu2 * log(eps) + (mu2 * n2) * (mu2 * n2) / 4));
  fprintf(stderr,"interp radius in USFFT: %d\n",m0);
  hipMalloc((void **)&f, n0 * n1 * n2 * sizeof(float2));
  hipMalloc((void **)&g, det * det * ntheta * sizeof(float2));
  hipMalloc((void **)&fdee,
             (2 * n0 + 2 * m0) * (2 * n1 + 2 * m1) * (2 * n2 + 2 * m2) * sizeof(float2));

  hipMalloc((void **)&x, det * det * ntheta * sizeof(float));
  hipMalloc((void **)&y, det * det * ntheta * sizeof(float));
  hipMalloc((void **)&z, det * det * ntheta * sizeof(float));
  hipMalloc((void **)&theta, ntheta * sizeof(float));
  
  int ffts[3];
  int idist;
  int inembed[3];
  // fft 2d
  ffts[0] = 2 * n2;
  ffts[1] = 2 * n1;
  ffts[2] = 2 * n0;
  idist = (2 * n0 + 2 * m0) * (2 * n1 + 2 * m1)* (2 * n2 + 2 * m2);
  inembed[0] = 2 * n2 + 2 * m2; // Note the order is reverse!
  inembed[1] = 2 * n1 + 2 * m1;
  inembed[2] = 2 * n0 + 2 * m0;
  hipfftPlanMany(&plan3d, 3, ffts, inembed, 1, idist, inembed, 1, idist,
                HIPFFT_C2C, 1);
  
  // fft 2d
  ffts[0] = det;
  ffts[1] = det;
  idist = det*det;
  inembed[0] = det;
  inembed[1] = det;
  hipfftPlanMany(&plan2d, 2, ffts, inembed, 1, idist, inembed, 1, idist,
                HIPFFT_C2C, ntheta);
  
  BS3d = dim3(16, 16, 4);

  GS3d0 = dim3(ceil(det / (float)BS3d.x), ceil(det / (float)BS3d.y),
                ceil(ntheta / (float)BS3d.z));
  GS3d1 = dim3(ceil(n0 / (float)BS3d.x), ceil(n1 / (float)BS3d.y),
                ceil(n2 / (float)BS3d.z));
  GS3d2 = dim3(ceil(2*n0 / (float)BS3d.x), 
               ceil(2*n1 / (float)BS3d.y),
               ceil(2*n2 / (float)BS3d.z));                
  GS3d3 = dim3(ceil((2 * n0 + 2 * m0) / (float)BS3d.x),
               ceil((2 * n1 + 2 * m1) / (float)BS3d.y), 
               ceil((2 * n2 + 2 * m2) / (float)BS3d.z));  
}

// destructor, memory deallocation
lamusfft::~lamusfft() { free(); }

void lamusfft::free() {
  if (!is_free) {
    hipFree(f);
    hipFree(g);
    hipFree(fdee);
    hipFree(x);
    hipFree(y);
    hipFree(z);
    hipfftDestroy(plan3d);
    hipfftDestroy(plan3d);
    hipfftDestroy(plan2d);
    is_free = true;
  }
}

void lamusfft::fwd(size_t g_, size_t f_, size_t theta_) {
  hipMemcpy(f, (float2 *)f_, n0 * n1 * n2 * sizeof(float2), hipMemcpyDefault);
  hipMemcpy(theta, (float *)theta_, ntheta * sizeof(float), hipMemcpyDefault);
  hipMemset(fdee, 0, (2 * n0 + 2 * m0) * (2 * n1 + 2 * m1) * (2 * n2 + 2 * m2) * sizeof(float2));

  takexyz <<<GS3d0, BS3d>>> (x, y, z, theta, phi, gamma, det, ntheta);

  divker <<<GS3d1, BS3d>>> (fdee, f, mu0, mu1, mu2, n0, n1, n2, m0, m1,m2, TOMO_FWD);  
  
  fftshiftc3d <<<GS3d3, BS3d>>> (fdee, 2 * n0 + 2 * m0, 2 * n1 +2 * m1, 2 * n2 +2 * m2);
  
  hipfftExecC2C(plan3d, (hipfftComplex *)&fdee[m0 + m1 * (2 * n0 + 2 * m0) + m2 * (2 * n0 + 2 * m0) * (2 * n1 + 2 * m1)].x,
                (hipfftComplex *)&fdee[m0 + m1 * (2 * n0 + 2 * m0) + m2 * (2 * n0 + 2 * m0) * (2 * n1 + 2 * m1)].x, HIPFFT_FORWARD);
  
  fftshiftc3d <<<GS3d3, BS3d>>> (fdee, 2 * n0 + 2 * m0, 2 * n1 +2 * m1, 2 * n2 +2 * m2);
  
  wrap <<<GS3d3, BS3d>>> (fdee, n0, n1, n2, m0, m1, m2, TOMO_FWD);
  gather <<<GS3d0, BS3d>>> (g, fdee, x, y, z, m0, m1, m2, mu0, mu1, mu2, n0, n1, n2, det, ntheta, TOMO_FWD);
  
  fftshiftc2d <<<GS3d0, BS3d>>> (g, det, ntheta);
  hipfftExecC2C(plan2d, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_BACKWARD);
  fftshiftc2d <<<GS3d0, BS3d>>> (g, det, ntheta);

  hipMemcpy((float2 *)g_, g, det * det * ntheta * sizeof(float2), hipMemcpyDefault);
}

void lamusfft::adj(size_t f_, size_t g_, size_t theta_) {
  hipMemcpy(g, (float2 *)g_, det * det * ntheta * sizeof(float2), hipMemcpyDefault);
  hipMemset(fdee, 0, (2 * n0 + 2 * m0) * (2 * n1 + 2 * m1) * (2 * n2 + 2 * m2) * sizeof(float2));

  takexyz <<<GS3d0, BS3d>>> (x, y, z, theta, phi, gamma, det, ntheta);

  fftshiftc2d <<<GS3d0, BS3d>>> (g, det, ntheta);
  hipfftExecC2C(plan2d, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_FORWARD);
  fftshiftc2d <<<GS3d0, BS3d>>> (g, det, ntheta);

  gather <<<GS3d0, BS3d>>> (g, fdee, x, y, z, m0, m1, m2, mu0, mu1, mu2, n0, n1, n2, det, ntheta, TOMO_ADJ);
  wrap <<<GS3d3, BS3d>>> (fdee, n0, n1, n2, m0, m1, m2, TOMO_ADJ);

  fftshiftc3d <<<GS3d3, BS3d>>> (fdee, 2 * n0 + 2 * m0, 2 * n1 +2 * m1, 2 * n2 +2 * m2);
  hipfftExecC2C(plan3d, (hipfftComplex *)&fdee[m0 + m1 * (2 * n0 + 2 * m0) + m2 * (2 * n0 + 2 * m0) * (2 * n1 + 2 * m1)],
                (hipfftComplex *)&fdee[m0 + m1 * (2 * n0 + 2 * m0) + m2 * (2 * n0 + 2 * m0) * (2 * n1 + 2 * m1)], HIPFFT_BACKWARD);
  fftshiftc3d <<<GS3d3, BS3d>>> (fdee, 2 * n0 + 2 * m0, 2 * n1 +2 * m1, 2 * n2 +2 * m2);
  

  divker <<<GS3d1, BS3d>>> (fdee, f, mu0, mu1, mu2, n0, n1, n2, m0,m1,m2, TOMO_ADJ);
  
  hipMemcpy((float2 *)f_, f, n0 * n1 * n2 * sizeof(float2),
              hipMemcpyDefault);
}
